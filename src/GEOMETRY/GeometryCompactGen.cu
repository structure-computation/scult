#include "hip/hip_runtime.h"
// generated file
// 
#include "GeometryCompactGen.h"

void launch_calculate_side_N_M_GPU(Ps<GeometryCompactClass> geometry_compact_CPU, Ps<GeometryCompactClass> geometry_compact_GPU){
    for(int i_group=0; i_group < geometry_compact_CPU->group_elements.size(); i_group++){
        int threadsPerBlock = 48;
        int sizeGrid = 6;
        int nGrids = std::floor( geometry_compact_CPU->group_elements[i_group].connectivities[0].size()  / (sizeGrid*threadsPerBlock) ) + 1 ;  // obtenir l'entier superieur
        int lastGridSize = (geometry_compact_CPU->group_elements[i_group].connectivities[0].size() - (sizeGrid*threadsPerBlock)*(nGrids-1))/threadsPerBlock;
        //PRINT(nGrids);
        //PRINT(lastGridSize);
        switch (geometry_compact_CPU->group_elements[i_group].pattern_id){
            //for pattern PATTERN_Triangle
            case 0 :{
                for(int i_grid=0; i_grid < nGrids; i_grid++){
                    int blocksPerGrid;
                    if(i_grid == nGrids-1){
                        blocksPerGrid = lastGridSize;
                    }else{
                        blocksPerGrid = sizeGrid;
                    }
                    //PRINT(i_grid);
                    //PRINT(blocksPerGrid);
                    //global_calculate_side_M_0<<<blocksPerGrid,threadsPerBlock>>>(geometry_compact_GPU.ptr(), i_group, i_grid, sizeGrid, geometry_compact_CPU->group_elements[i_group].nb_elements);
                    global_calculate_side_N_0<<<blocksPerGrid,threadsPerBlock>>>(geometry_compact_GPU.ptr(), i_group, i_grid, sizeGrid, geometry_compact_CPU->group_elements[i_group].nb_elements);
                }
                break;
            }
            //for pattern PATTERN_Triangle_6
            case 1 :{
                for(int i_grid=0; i_grid < nGrids; i_grid++){
                    int blocksPerGrid;
                    if(i_grid == nGrids-1){
                        blocksPerGrid = lastGridSize;
                    }else{
                        blocksPerGrid = sizeGrid;
                    }
                    //PRINT(i_grid);
                    //PRINT(blocksPerGrid);
                    //global_calculate_side_M_1<<<blocksPerGrid,threadsPerBlock>>>(geometry_compact_GPU.ptr(), i_group, i_grid, sizeGrid, geometry_compact_CPU->group_elements[i_group].nb_elements);
                    global_calculate_side_N_1<<<blocksPerGrid,threadsPerBlock>>>(geometry_compact_GPU.ptr(), i_group, i_grid, sizeGrid, geometry_compact_CPU->group_elements[i_group].nb_elements);
                }
                break;
            }
            //for pattern PATTERN_Tetra
            case 2 :{
                for(int i_grid=0; i_grid < nGrids; i_grid++){
                    int blocksPerGrid;
                    if(i_grid == nGrids-1){
                        blocksPerGrid = lastGridSize;
                    }else{
                        blocksPerGrid = sizeGrid;
                    }
                    //PRINT(i_grid);
                    //PRINT(blocksPerGrid);
                    //global_calculate_side_M_2<<<blocksPerGrid,threadsPerBlock>>>(geometry_compact_GPU.ptr(), i_group, i_grid, sizeGrid, geometry_compact_CPU->group_elements[i_group].nb_elements);
                    global_calculate_side_N_2<<<blocksPerGrid,threadsPerBlock>>>(geometry_compact_GPU.ptr(), i_group, i_grid, sizeGrid, geometry_compact_CPU->group_elements[i_group].nb_elements);
                }
                break;
            }
            //for pattern PATTERN_Tetra_10
            case 3 :{
                for(int i_grid=0; i_grid < nGrids; i_grid++){
                    int blocksPerGrid;
                    if(i_grid == nGrids-1){
                        blocksPerGrid = lastGridSize;
                    }else{
                        blocksPerGrid = sizeGrid;
                    }
                    //PRINT(i_grid);
                    //PRINT(blocksPerGrid);
                    //global_calculate_side_M_3<<<blocksPerGrid,threadsPerBlock>>>(geometry_compact_GPU.ptr(), i_group, i_grid, sizeGrid, geometry_compact_CPU->group_elements[i_group].nb_elements);
                    global_calculate_side_N_3<<<blocksPerGrid,threadsPerBlock>>>(geometry_compact_GPU.ptr(), i_group, i_grid, sizeGrid, geometry_compact_CPU->group_elements[i_group].nb_elements);
                }
                break;
            }
            default :{
                std::cerr << "type de pattern non implemente" << std::endl; assert(0);                    
            }
        }
    }
}

void launch_calculate_side_N_M_CPU(Ps<GeometryCompactClass> geometry_compact_CPU){
    for(int i_group=0; i_group < geometry_compact_CPU->group_elements.size(); i_group++){
        switch (geometry_compact_CPU->group_elements[i_group].pattern_id){
            //for pattern PATTERN_Triangle
            case 0 :{
                //global_calculate_side_M_0_CPU(geometry_compact_CPU.ptr(), i_group);
                global_calculate_side_N_0_CPU(geometry_compact_CPU.ptr(), i_group);
                break;
            }
            //for pattern PATTERN_Triangle_6
            case 1 :{
                //global_calculate_side_M_1_CPU(geometry_compact_CPU.ptr(), i_group);
                global_calculate_side_N_1_CPU(geometry_compact_CPU.ptr(), i_group);
                break;
            }
            //for pattern PATTERN_Tetra
            case 2 :{
                //global_calculate_side_M_2_CPU(geometry_compact_CPU.ptr(), i_group);
                global_calculate_side_N_2_CPU(geometry_compact_CPU.ptr(), i_group);
                break;
            }
            //for pattern PATTERN_Tetra_10
            case 3 :{
                //global_calculate_side_M_3_CPU(geometry_compact_CPU.ptr(), i_group);
                global_calculate_side_N_3_CPU(geometry_compact_CPU.ptr(), i_group);
                break;
            }
            default :{
                std::cerr << "type de pattern non implemente" << std::endl; assert(0);
            }
            
        }
    }
}

void launch_calculate_interface_N_M_CPU(Ps<GeometryCompactClass> geometry_compact_CPU){
    for(int i_group=0; i_group < geometry_compact_CPU->group_interfaces.size(); i_group++){
        switch (geometry_compact_CPU->group_interfaces[i_group].patterns_id[0]){
            //for pattern Bar
            case 0 :{
                global_calculate_interface_NeMe_0_CPU(geometry_compact_CPU.ptr(), i_group);
                break;
            }
            //for pattern Bar_3
            case 1 :{
                global_calculate_interface_NeMe_1_CPU(geometry_compact_CPU.ptr(), i_group);
                break;
            }
            //for pattern Triangle
            case 2 :{
                global_calculate_interface_NeMe_2_CPU(geometry_compact_CPU.ptr(), i_group);
                break;
            }
            //for pattern Triangle_6
            case 3 :{
                global_calculate_interface_NeMe_3_CPU(geometry_compact_CPU.ptr(), i_group);
                break;
            }
            default :{
                std::cerr << "type de pattern non implemente" << std::endl; assert(0);
            }
            
        }
    }
}

