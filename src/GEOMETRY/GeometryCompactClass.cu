#include "hip/hip_runtime.h"
#include "GeometryCompactClass.h"

BEG_METIL_NAMESPACE

void GeometryCompactClass::update_ptr_cpu_load( ST off ) {
    (char *&)patterns.types.data_ += off;
    for( ST i = 0; i < patterns.types.size_; ++i ) {
        (char *&)patterns.types.data_[ i ].permutation.data_ += off;
        (char *&)patterns.types.data_[ i ].sides.data_ += off;
        for( ST j = 0; j < patterns.types.data_[ i ].sides.size_; ++j ) {
            (char *&)patterns.types.data_[ i ].sides.data_[ j ].data_ += off;
        }
    }
    (char *&)mesh_nodes.data_ += off;
    for( ST i = 0; i < mesh_nodes.size_; ++i ) {
        (char *&)mesh_nodes.data_[ i ].data_ += off;
    }
    (char *&)nodes.data_ += off;
    for( ST i = 0; i < nodes.size_; ++i ) {
        (char *&)nodes.data_[ i ].data_ += off;
    }
    (char *&)group_elements.data_ += off;
    for( ST i = 0; i < group_elements.size_; ++i ) {
        (char *&)group_elements.data_[ i ].mesh_connectivities.data_ += off;
        for( ST j = 0; j < group_elements.data_[ i ].mesh_connectivities.size_; ++j ) {
            (char *&)group_elements.data_[ i ].mesh_connectivities.data_[ j ].data_ += off;
        }
        (char *&)group_elements.data_[ i ].connectivities.data_ += off;
        for( ST j = 0; j < group_elements.data_[ i ].connectivities.size_; ++j ) {
            (char *&)group_elements.data_[ i ].connectivities.data_[ j ].data_ += off;
        }
        (char *&)group_elements.data_[ i ].interface_group_id.data_ += off;
        for( ST j = 0; j < group_elements.data_[ i ].interface_group_id.size_; ++j ) {
            (char *&)group_elements.data_[ i ].interface_group_id.data_[ j ].data_ += off;
        }
        (char *&)group_elements.data_[ i ].interface_num_in_group.data_ += off;
        for( ST j = 0; j < group_elements.data_[ i ].interface_num_in_group.size_; ++j ) {
            (char *&)group_elements.data_[ i ].interface_num_in_group.data_[ j ].data_ += off;
        }
        (char *&)group_elements.data_[ i ].group_interfaces_id.data_ += off;
        for( ST j = 0; j < group_elements.data_[ i ].group_interfaces_id.size_; ++j ) {
            (char *&)group_elements.data_[ i ].group_interfaces_id.data_[ j ].data_ += off;
        }
        (char *&)group_elements.data_[ i ].side_N.data_ += off;
        for( ST j = 0; j < group_elements.data_[ i ].side_N.size_; ++j ) {
            (char *&)group_elements.data_[ i ].side_N.data_[ j ].data_ += off;
            for( ST k = 0; k < group_elements.data_[ i ].side_N.data_[ j ].size_; ++k ) {
                (char *&)group_elements.data_[ i ].side_N.data_[ j ].data_[ k ].data_ += off;
                for( ST l = 0; l < group_elements.data_[ i ].side_N.data_[ j ].data_[ k ].size_; ++l ) {
                    (char *&)group_elements.data_[ i ].side_N.data_[ j ].data_[ k ].data_[ l ].data_ += off;
                }
            }
        }
        (char *&)group_elements.data_[ i ].side_M.data_ += off;
        for( ST j = 0; j < group_elements.data_[ i ].side_M.size_; ++j ) {
            (char *&)group_elements.data_[ i ].side_M.data_[ j ].data_ += off;
            for( ST k = 0; k < group_elements.data_[ i ].side_M.data_[ j ].size_; ++k ) {
                (char *&)group_elements.data_[ i ].side_M.data_[ j ].data_[ k ].data_ += off;
            }
        }
    }
    (char *&)group_interfaces.data_ += off;
    for( ST i = 0; i < group_interfaces.size_; ++i ) {
        (char *&)group_interfaces.data_[ i ].group_elements_id.data_ += off;
        (char *&)group_interfaces.data_[ i ].patterns_id.data_ += off;
        (char *&)group_interfaces.data_[ i ].element_num_in_group.data_ += off;
        for( ST j = 0; j < group_interfaces.data_[ i ].element_num_in_group.size_; ++j ) {
            (char *&)group_interfaces.data_[ i ].element_num_in_group.data_[ j ].data_ += off;
        }
        (char *&)group_interfaces.data_[ i ].element_num_side.data_ += off;
        for( ST j = 0; j < group_interfaces.data_[ i ].element_num_side.size_; ++j ) {
            (char *&)group_interfaces.data_[ i ].element_num_side.data_[ j ].data_ += off;
        }
        (char *&)group_interfaces.data_[ i ].mesh_connectivities.data_ += off;
        for( ST j = 0; j < group_interfaces.data_[ i ].mesh_connectivities.size_; ++j ) {
            (char *&)group_interfaces.data_[ i ].mesh_connectivities.data_[ j ].data_ += off;
        }
        (char *&)group_interfaces.data_[ i ].connectivities.data_ += off;
        for( ST j = 0; j < group_interfaces.data_[ i ].connectivities.size_; ++j ) {
            (char *&)group_interfaces.data_[ i ].connectivities.data_[ j ].data_ += off;
        }
        (char *&)group_interfaces.data_[ i ].correspondance_between_nodes.data_ += off;
        for( ST j = 0; j < group_interfaces.data_[ i ].correspondance_between_nodes.size_; ++j ) {
            (char *&)group_interfaces.data_[ i ].correspondance_between_nodes.data_[ j ].data_ += off;
        }
        (char *&)group_interfaces.data_[ i ].Ne.data_ += off;
        for( ST j = 0; j < group_interfaces.data_[ i ].Ne.size_; ++j ) {
            (char *&)group_interfaces.data_[ i ].Ne.data_[ j ].data_ += off;
        }
        (char *&)group_interfaces.data_[ i ].Me.data_ += off;
    }
}

__global__
void GeometryCompactClass__update_ptr_gpu_load( GeometryCompactClass *obj, ST off ) {
    (char *&)obj->patterns.types.data_ += off;
    for( ST i = 0; i < obj->patterns.types.size_; ++i ) {
        (char *&)obj->patterns.types.data_[ i ].permutation.data_ += off;
        (char *&)obj->patterns.types.data_[ i ].sides.data_ += off;
        for( ST j = 0; j < obj->patterns.types.data_[ i ].sides.size_; ++j ) {
            (char *&)obj->patterns.types.data_[ i ].sides.data_[ j ].data_ += off;
        }
    }
    (char *&)obj->mesh_nodes.data_ += off;
    for( ST i = 0; i < obj->mesh_nodes.size_; ++i ) {
        (char *&)obj->mesh_nodes.data_[ i ].data_ += off;
    }
    (char *&)obj->nodes.data_ += off;
    for( ST i = 0; i < obj->nodes.size_; ++i ) {
        (char *&)obj->nodes.data_[ i ].data_ += off;
    }
    (char *&)obj->group_elements.data_ += off;
    for( ST i = 0; i < obj->group_elements.size_; ++i ) {
        (char *&)obj->group_elements.data_[ i ].mesh_connectivities.data_ += off;
        for( ST j = 0; j < obj->group_elements.data_[ i ].mesh_connectivities.size_; ++j ) {
            (char *&)obj->group_elements.data_[ i ].mesh_connectivities.data_[ j ].data_ += off;
        }
        (char *&)obj->group_elements.data_[ i ].connectivities.data_ += off;
        for( ST j = 0; j < obj->group_elements.data_[ i ].connectivities.size_; ++j ) {
            (char *&)obj->group_elements.data_[ i ].connectivities.data_[ j ].data_ += off;
        }
        (char *&)obj->group_elements.data_[ i ].interface_group_id.data_ += off;
        for( ST j = 0; j < obj->group_elements.data_[ i ].interface_group_id.size_; ++j ) {
            (char *&)obj->group_elements.data_[ i ].interface_group_id.data_[ j ].data_ += off;
        }
        (char *&)obj->group_elements.data_[ i ].interface_num_in_group.data_ += off;
        for( ST j = 0; j < obj->group_elements.data_[ i ].interface_num_in_group.size_; ++j ) {
            (char *&)obj->group_elements.data_[ i ].interface_num_in_group.data_[ j ].data_ += off;
        }
        (char *&)obj->group_elements.data_[ i ].group_interfaces_id.data_ += off;
        for( ST j = 0; j < obj->group_elements.data_[ i ].group_interfaces_id.size_; ++j ) {
            (char *&)obj->group_elements.data_[ i ].group_interfaces_id.data_[ j ].data_ += off;
        }
        (char *&)obj->group_elements.data_[ i ].side_N.data_ += off;
        for( ST j = 0; j < obj->group_elements.data_[ i ].side_N.size_; ++j ) {
            (char *&)obj->group_elements.data_[ i ].side_N.data_[ j ].data_ += off;
            for( ST k = 0; k < obj->group_elements.data_[ i ].side_N.data_[ j ].size_; ++k ) {
                (char *&)obj->group_elements.data_[ i ].side_N.data_[ j ].data_[ k ].data_ += off;
                for( ST l = 0; l < obj->group_elements.data_[ i ].side_N.data_[ j ].data_[ k ].size_; ++l ) {
                    (char *&)obj->group_elements.data_[ i ].side_N.data_[ j ].data_[ k ].data_[ l ].data_ += off;
                }
            }
        }
        (char *&)obj->group_elements.data_[ i ].side_M.data_ += off;
        for( ST j = 0; j < obj->group_elements.data_[ i ].side_M.size_; ++j ) {
            (char *&)obj->group_elements.data_[ i ].side_M.data_[ j ].data_ += off;
            for( ST k = 0; k < obj->group_elements.data_[ i ].side_M.data_[ j ].size_; ++k ) {
                (char *&)obj->group_elements.data_[ i ].side_M.data_[ j ].data_[ k ].data_ += off;
            }
        }
    }
    (char *&)obj->group_interfaces.data_ += off;
    for( ST i = 0; i < obj->group_interfaces.size_; ++i ) {
        (char *&)obj->group_interfaces.data_[ i ].group_elements_id.data_ += off;
        (char *&)obj->group_interfaces.data_[ i ].patterns_id.data_ += off;
        (char *&)obj->group_interfaces.data_[ i ].element_num_in_group.data_ += off;
        for( ST j = 0; j < obj->group_interfaces.data_[ i ].element_num_in_group.size_; ++j ) {
            (char *&)obj->group_interfaces.data_[ i ].element_num_in_group.data_[ j ].data_ += off;
        }
        (char *&)obj->group_interfaces.data_[ i ].element_num_side.data_ += off;
        for( ST j = 0; j < obj->group_interfaces.data_[ i ].element_num_side.size_; ++j ) {
            (char *&)obj->group_interfaces.data_[ i ].element_num_side.data_[ j ].data_ += off;
        }
        (char *&)obj->group_interfaces.data_[ i ].mesh_connectivities.data_ += off;
        for( ST j = 0; j < obj->group_interfaces.data_[ i ].mesh_connectivities.size_; ++j ) {
            (char *&)obj->group_interfaces.data_[ i ].mesh_connectivities.data_[ j ].data_ += off;
        }
        (char *&)obj->group_interfaces.data_[ i ].connectivities.data_ += off;
        for( ST j = 0; j < obj->group_interfaces.data_[ i ].connectivities.size_; ++j ) {
            (char *&)obj->group_interfaces.data_[ i ].connectivities.data_[ j ].data_ += off;
        }
        (char *&)obj->group_interfaces.data_[ i ].correspondance_between_nodes.data_ += off;
        for( ST j = 0; j < obj->group_interfaces.data_[ i ].correspondance_between_nodes.size_; ++j ) {
            (char *&)obj->group_interfaces.data_[ i ].correspondance_between_nodes.data_[ j ].data_ += off;
        }
        (char *&)obj->group_interfaces.data_[ i ].Ne.data_ += off;
        for( ST j = 0; j < obj->group_interfaces.data_[ i ].Ne.size_; ++j ) {
            (char *&)obj->group_interfaces.data_[ i ].Ne.data_[ j ].data_ += off;
        }
        (char *&)obj->group_interfaces.data_[ i ].Me.data_ += off;
    }
}

void GeometryCompactClass::update_ptr_gpu_load( ST off ) {
    GeometryCompactClass__update_ptr_gpu_load<<<1,1>>>( this, off );
}

void GeometryCompactClass::update_ptr_cpu_save( ST off ) {
    for( ST i = 0; i < patterns.types.size_; ++i ) {
        (char *&)patterns.types.data_[ i ].permutation.data_ += off;
        for( ST j = 0; j < patterns.types.data_[ i ].sides.size_; ++j ) {
            (char *&)patterns.types.data_[ i ].sides.data_[ j ].data_ += off;
        }
        (char *&)patterns.types.data_[ i ].sides.data_ += off;
    }
    (char *&)patterns.types.data_ += off;
    for( ST i = 0; i < mesh_nodes.size_; ++i ) {
        (char *&)mesh_nodes.data_[ i ].data_ += off;
    }
    (char *&)mesh_nodes.data_ += off;
    for( ST i = 0; i < nodes.size_; ++i ) {
        (char *&)nodes.data_[ i ].data_ += off;
    }
    (char *&)nodes.data_ += off;
    for( ST i = 0; i < group_elements.size_; ++i ) {
        for( ST j = 0; j < group_elements.data_[ i ].mesh_connectivities.size_; ++j ) {
            (char *&)group_elements.data_[ i ].mesh_connectivities.data_[ j ].data_ += off;
        }
        (char *&)group_elements.data_[ i ].mesh_connectivities.data_ += off;
        for( ST j = 0; j < group_elements.data_[ i ].connectivities.size_; ++j ) {
            (char *&)group_elements.data_[ i ].connectivities.data_[ j ].data_ += off;
        }
        (char *&)group_elements.data_[ i ].connectivities.data_ += off;
        for( ST j = 0; j < group_elements.data_[ i ].interface_group_id.size_; ++j ) {
            (char *&)group_elements.data_[ i ].interface_group_id.data_[ j ].data_ += off;
        }
        (char *&)group_elements.data_[ i ].interface_group_id.data_ += off;
        for( ST j = 0; j < group_elements.data_[ i ].interface_num_in_group.size_; ++j ) {
            (char *&)group_elements.data_[ i ].interface_num_in_group.data_[ j ].data_ += off;
        }
        (char *&)group_elements.data_[ i ].interface_num_in_group.data_ += off;
        for( ST j = 0; j < group_elements.data_[ i ].group_interfaces_id.size_; ++j ) {
            (char *&)group_elements.data_[ i ].group_interfaces_id.data_[ j ].data_ += off;
        }
        (char *&)group_elements.data_[ i ].group_interfaces_id.data_ += off;
        for( ST j = 0; j < group_elements.data_[ i ].side_N.size_; ++j ) {
            for( ST k = 0; k < group_elements.data_[ i ].side_N.data_[ j ].size_; ++k ) {
                for( ST l = 0; l < group_elements.data_[ i ].side_N.data_[ j ].data_[ k ].size_; ++l ) {
                    (char *&)group_elements.data_[ i ].side_N.data_[ j ].data_[ k ].data_[ l ].data_ += off;
                }
                (char *&)group_elements.data_[ i ].side_N.data_[ j ].data_[ k ].data_ += off;
            }
            (char *&)group_elements.data_[ i ].side_N.data_[ j ].data_ += off;
        }
        (char *&)group_elements.data_[ i ].side_N.data_ += off;
        for( ST j = 0; j < group_elements.data_[ i ].side_M.size_; ++j ) {
            for( ST k = 0; k < group_elements.data_[ i ].side_M.data_[ j ].size_; ++k ) {
                (char *&)group_elements.data_[ i ].side_M.data_[ j ].data_[ k ].data_ += off;
            }
            (char *&)group_elements.data_[ i ].side_M.data_[ j ].data_ += off;
        }
        (char *&)group_elements.data_[ i ].side_M.data_ += off;
    }
    (char *&)group_elements.data_ += off;
    for( ST i = 0; i < group_interfaces.size_; ++i ) {
        (char *&)group_interfaces.data_[ i ].group_elements_id.data_ += off;
        (char *&)group_interfaces.data_[ i ].patterns_id.data_ += off;
        for( ST j = 0; j < group_interfaces.data_[ i ].element_num_in_group.size_; ++j ) {
            (char *&)group_interfaces.data_[ i ].element_num_in_group.data_[ j ].data_ += off;
        }
        (char *&)group_interfaces.data_[ i ].element_num_in_group.data_ += off;
        for( ST j = 0; j < group_interfaces.data_[ i ].element_num_side.size_; ++j ) {
            (char *&)group_interfaces.data_[ i ].element_num_side.data_[ j ].data_ += off;
        }
        (char *&)group_interfaces.data_[ i ].element_num_side.data_ += off;
        for( ST j = 0; j < group_interfaces.data_[ i ].mesh_connectivities.size_; ++j ) {
            (char *&)group_interfaces.data_[ i ].mesh_connectivities.data_[ j ].data_ += off;
        }
        (char *&)group_interfaces.data_[ i ].mesh_connectivities.data_ += off;
        for( ST j = 0; j < group_interfaces.data_[ i ].connectivities.size_; ++j ) {
            (char *&)group_interfaces.data_[ i ].connectivities.data_[ j ].data_ += off;
        }
        (char *&)group_interfaces.data_[ i ].connectivities.data_ += off;
        for( ST j = 0; j < group_interfaces.data_[ i ].correspondance_between_nodes.size_; ++j ) {
            (char *&)group_interfaces.data_[ i ].correspondance_between_nodes.data_[ j ].data_ += off;
        }
        (char *&)group_interfaces.data_[ i ].correspondance_between_nodes.data_ += off;
        for( ST j = 0; j < group_interfaces.data_[ i ].Ne.size_; ++j ) {
            (char *&)group_interfaces.data_[ i ].Ne.data_[ j ].data_ += off;
        }
        (char *&)group_interfaces.data_[ i ].Ne.data_ += off;
        (char *&)group_interfaces.data_[ i ].Me.data_ += off;
    }
    (char *&)group_interfaces.data_ += off;
}

__global__
void GeometryCompactClass__update_ptr_gpu_save( GeometryCompactClass *obj, ST off ) {
    for( ST i = 0; i < obj->patterns.types.size_; ++i ) {
        (char *&)obj->patterns.types.data_[ i ].permutation.data_ += off;
        for( ST j = 0; j < obj->patterns.types.data_[ i ].sides.size_; ++j ) {
            (char *&)obj->patterns.types.data_[ i ].sides.data_[ j ].data_ += off;
        }
        (char *&)obj->patterns.types.data_[ i ].sides.data_ += off;
    }
    (char *&)obj->patterns.types.data_ += off;
    for( ST i = 0; i < obj->mesh_nodes.size_; ++i ) {
        (char *&)obj->mesh_nodes.data_[ i ].data_ += off;
    }
    (char *&)obj->mesh_nodes.data_ += off;
    for( ST i = 0; i < obj->nodes.size_; ++i ) {
        (char *&)obj->nodes.data_[ i ].data_ += off;
    }
    (char *&)obj->nodes.data_ += off;
    for( ST i = 0; i < obj->group_elements.size_; ++i ) {
        for( ST j = 0; j < obj->group_elements.data_[ i ].mesh_connectivities.size_; ++j ) {
            (char *&)obj->group_elements.data_[ i ].mesh_connectivities.data_[ j ].data_ += off;
        }
        (char *&)obj->group_elements.data_[ i ].mesh_connectivities.data_ += off;
        for( ST j = 0; j < obj->group_elements.data_[ i ].connectivities.size_; ++j ) {
            (char *&)obj->group_elements.data_[ i ].connectivities.data_[ j ].data_ += off;
        }
        (char *&)obj->group_elements.data_[ i ].connectivities.data_ += off;
        for( ST j = 0; j < obj->group_elements.data_[ i ].interface_group_id.size_; ++j ) {
            (char *&)obj->group_elements.data_[ i ].interface_group_id.data_[ j ].data_ += off;
        }
        (char *&)obj->group_elements.data_[ i ].interface_group_id.data_ += off;
        for( ST j = 0; j < obj->group_elements.data_[ i ].interface_num_in_group.size_; ++j ) {
            (char *&)obj->group_elements.data_[ i ].interface_num_in_group.data_[ j ].data_ += off;
        }
        (char *&)obj->group_elements.data_[ i ].interface_num_in_group.data_ += off;
        for( ST j = 0; j < obj->group_elements.data_[ i ].group_interfaces_id.size_; ++j ) {
            (char *&)obj->group_elements.data_[ i ].group_interfaces_id.data_[ j ].data_ += off;
        }
        (char *&)obj->group_elements.data_[ i ].group_interfaces_id.data_ += off;
        for( ST j = 0; j < obj->group_elements.data_[ i ].side_N.size_; ++j ) {
            for( ST k = 0; k < obj->group_elements.data_[ i ].side_N.data_[ j ].size_; ++k ) {
                for( ST l = 0; l < obj->group_elements.data_[ i ].side_N.data_[ j ].data_[ k ].size_; ++l ) {
                    (char *&)obj->group_elements.data_[ i ].side_N.data_[ j ].data_[ k ].data_[ l ].data_ += off;
                }
                (char *&)obj->group_elements.data_[ i ].side_N.data_[ j ].data_[ k ].data_ += off;
            }
            (char *&)obj->group_elements.data_[ i ].side_N.data_[ j ].data_ += off;
        }
        (char *&)obj->group_elements.data_[ i ].side_N.data_ += off;
        for( ST j = 0; j < obj->group_elements.data_[ i ].side_M.size_; ++j ) {
            for( ST k = 0; k < obj->group_elements.data_[ i ].side_M.data_[ j ].size_; ++k ) {
                (char *&)obj->group_elements.data_[ i ].side_M.data_[ j ].data_[ k ].data_ += off;
            }
            (char *&)obj->group_elements.data_[ i ].side_M.data_[ j ].data_ += off;
        }
        (char *&)obj->group_elements.data_[ i ].side_M.data_ += off;
    }
    (char *&)obj->group_elements.data_ += off;
    for( ST i = 0; i < obj->group_interfaces.size_; ++i ) {
        (char *&)obj->group_interfaces.data_[ i ].group_elements_id.data_ += off;
        (char *&)obj->group_interfaces.data_[ i ].patterns_id.data_ += off;
        for( ST j = 0; j < obj->group_interfaces.data_[ i ].element_num_in_group.size_; ++j ) {
            (char *&)obj->group_interfaces.data_[ i ].element_num_in_group.data_[ j ].data_ += off;
        }
        (char *&)obj->group_interfaces.data_[ i ].element_num_in_group.data_ += off;
        for( ST j = 0; j < obj->group_interfaces.data_[ i ].element_num_side.size_; ++j ) {
            (char *&)obj->group_interfaces.data_[ i ].element_num_side.data_[ j ].data_ += off;
        }
        (char *&)obj->group_interfaces.data_[ i ].element_num_side.data_ += off;
        for( ST j = 0; j < obj->group_interfaces.data_[ i ].mesh_connectivities.size_; ++j ) {
            (char *&)obj->group_interfaces.data_[ i ].mesh_connectivities.data_[ j ].data_ += off;
        }
        (char *&)obj->group_interfaces.data_[ i ].mesh_connectivities.data_ += off;
        for( ST j = 0; j < obj->group_interfaces.data_[ i ].connectivities.size_; ++j ) {
            (char *&)obj->group_interfaces.data_[ i ].connectivities.data_[ j ].data_ += off;
        }
        (char *&)obj->group_interfaces.data_[ i ].connectivities.data_ += off;
        for( ST j = 0; j < obj->group_interfaces.data_[ i ].correspondance_between_nodes.size_; ++j ) {
            (char *&)obj->group_interfaces.data_[ i ].correspondance_between_nodes.data_[ j ].data_ += off;
        }
        (char *&)obj->group_interfaces.data_[ i ].correspondance_between_nodes.data_ += off;
        for( ST j = 0; j < obj->group_interfaces.data_[ i ].Ne.size_; ++j ) {
            (char *&)obj->group_interfaces.data_[ i ].Ne.data_[ j ].data_ += off;
        }
        (char *&)obj->group_interfaces.data_[ i ].Ne.data_ += off;
        (char *&)obj->group_interfaces.data_[ i ].Me.data_ += off;
    }
    (char *&)obj->group_interfaces.data_ += off;
}

void GeometryCompactClass::update_ptr_gpu_save( ST off ) {
    GeometryCompactClass__update_ptr_gpu_save<<<1,1>>>( this, off );
}

void GeometryCompactClass::Patterns::update_ptr_cpu_load( ST off ) {
    (char *&)types.data_ += off;
    for( ST i = 0; i < types.size_; ++i ) {
        (char *&)types.data_[ i ].permutation.data_ += off;
        (char *&)types.data_[ i ].sides.data_ += off;
        for( ST j = 0; j < types.data_[ i ].sides.size_; ++j ) {
            (char *&)types.data_[ i ].sides.data_[ j ].data_ += off;
        }
    }
}

__global__
void GeometryCompactClass__Patterns__update_ptr_gpu_load( GeometryCompactClass::Patterns *obj, ST off ) {
    (char *&)obj->types.data_ += off;
    for( ST i = 0; i < obj->types.size_; ++i ) {
        (char *&)obj->types.data_[ i ].permutation.data_ += off;
        (char *&)obj->types.data_[ i ].sides.data_ += off;
        for( ST j = 0; j < obj->types.data_[ i ].sides.size_; ++j ) {
            (char *&)obj->types.data_[ i ].sides.data_[ j ].data_ += off;
        }
    }
}

void GeometryCompactClass::Patterns::update_ptr_gpu_load( ST off ) {
    GeometryCompactClass__Patterns__update_ptr_gpu_load<<<1,1>>>( this, off );
}

void GeometryCompactClass::Patterns::update_ptr_cpu_save( ST off ) {
    for( ST i = 0; i < types.size_; ++i ) {
        (char *&)types.data_[ i ].permutation.data_ += off;
        for( ST j = 0; j < types.data_[ i ].sides.size_; ++j ) {
            (char *&)types.data_[ i ].sides.data_[ j ].data_ += off;
        }
        (char *&)types.data_[ i ].sides.data_ += off;
    }
    (char *&)types.data_ += off;
}

__global__
void GeometryCompactClass__Patterns__update_ptr_gpu_save( GeometryCompactClass::Patterns *obj, ST off ) {
    for( ST i = 0; i < obj->types.size_; ++i ) {
        (char *&)obj->types.data_[ i ].permutation.data_ += off;
        for( ST j = 0; j < obj->types.data_[ i ].sides.size_; ++j ) {
            (char *&)obj->types.data_[ i ].sides.data_[ j ].data_ += off;
        }
        (char *&)obj->types.data_[ i ].sides.data_ += off;
    }
    (char *&)obj->types.data_ += off;
}

void GeometryCompactClass::Patterns::update_ptr_gpu_save( ST off ) {
    GeometryCompactClass__Patterns__update_ptr_gpu_save<<<1,1>>>( this, off );
}

void GeometryCompactClass::Patterns::Types::update_ptr_cpu_load( ST off ) {
    (char *&)permutation.data_ += off;
    (char *&)sides.data_ += off;
    for( ST i = 0; i < sides.size_; ++i ) {
        (char *&)sides.data_[ i ].data_ += off;
    }
}

__global__
void GeometryCompactClass__Patterns__Types__update_ptr_gpu_load( GeometryCompactClass::Patterns::Types *obj, ST off ) {
    (char *&)obj->permutation.data_ += off;
    (char *&)obj->sides.data_ += off;
    for( ST i = 0; i < obj->sides.size_; ++i ) {
        (char *&)obj->sides.data_[ i ].data_ += off;
    }
}

void GeometryCompactClass::Patterns::Types::update_ptr_gpu_load( ST off ) {
    GeometryCompactClass__Patterns__Types__update_ptr_gpu_load<<<1,1>>>( this, off );
}

void GeometryCompactClass::Patterns::Types::update_ptr_cpu_save( ST off ) {
    (char *&)permutation.data_ += off;
    for( ST i = 0; i < sides.size_; ++i ) {
        (char *&)sides.data_[ i ].data_ += off;
    }
    (char *&)sides.data_ += off;
}

__global__
void GeometryCompactClass__Patterns__Types__update_ptr_gpu_save( GeometryCompactClass::Patterns::Types *obj, ST off ) {
    (char *&)obj->permutation.data_ += off;
    for( ST i = 0; i < obj->sides.size_; ++i ) {
        (char *&)obj->sides.data_[ i ].data_ += off;
    }
    (char *&)obj->sides.data_ += off;
}

void GeometryCompactClass::Patterns::Types::update_ptr_gpu_save( ST off ) {
    GeometryCompactClass__Patterns__Types__update_ptr_gpu_save<<<1,1>>>( this, off );
}

void GeometryCompactClass::GroupElementsCompactClass::update_ptr_cpu_load( ST off ) {
    (char *&)mesh_connectivities.data_ += off;
    for( ST i = 0; i < mesh_connectivities.size_; ++i ) {
        (char *&)mesh_connectivities.data_[ i ].data_ += off;
    }
    (char *&)connectivities.data_ += off;
    for( ST i = 0; i < connectivities.size_; ++i ) {
        (char *&)connectivities.data_[ i ].data_ += off;
    }
    (char *&)interface_group_id.data_ += off;
    for( ST i = 0; i < interface_group_id.size_; ++i ) {
        (char *&)interface_group_id.data_[ i ].data_ += off;
    }
    (char *&)interface_num_in_group.data_ += off;
    for( ST i = 0; i < interface_num_in_group.size_; ++i ) {
        (char *&)interface_num_in_group.data_[ i ].data_ += off;
    }
    (char *&)group_interfaces_id.data_ += off;
    for( ST i = 0; i < group_interfaces_id.size_; ++i ) {
        (char *&)group_interfaces_id.data_[ i ].data_ += off;
    }
    (char *&)side_N.data_ += off;
    for( ST i = 0; i < side_N.size_; ++i ) {
        (char *&)side_N.data_[ i ].data_ += off;
        for( ST j = 0; j < side_N.data_[ i ].size_; ++j ) {
            (char *&)side_N.data_[ i ].data_[ j ].data_ += off;
            for( ST k = 0; k < side_N.data_[ i ].data_[ j ].size_; ++k ) {
                (char *&)side_N.data_[ i ].data_[ j ].data_[ k ].data_ += off;
            }
        }
    }
    (char *&)side_M.data_ += off;
    for( ST i = 0; i < side_M.size_; ++i ) {
        (char *&)side_M.data_[ i ].data_ += off;
        for( ST j = 0; j < side_M.data_[ i ].size_; ++j ) {
            (char *&)side_M.data_[ i ].data_[ j ].data_ += off;
        }
    }
}

__global__
void GeometryCompactClass__GroupElementsCompactClass__update_ptr_gpu_load( GeometryCompactClass::GroupElementsCompactClass *obj, ST off ) {
    (char *&)obj->mesh_connectivities.data_ += off;
    for( ST i = 0; i < obj->mesh_connectivities.size_; ++i ) {
        (char *&)obj->mesh_connectivities.data_[ i ].data_ += off;
    }
    (char *&)obj->connectivities.data_ += off;
    for( ST i = 0; i < obj->connectivities.size_; ++i ) {
        (char *&)obj->connectivities.data_[ i ].data_ += off;
    }
    (char *&)obj->interface_group_id.data_ += off;
    for( ST i = 0; i < obj->interface_group_id.size_; ++i ) {
        (char *&)obj->interface_group_id.data_[ i ].data_ += off;
    }
    (char *&)obj->interface_num_in_group.data_ += off;
    for( ST i = 0; i < obj->interface_num_in_group.size_; ++i ) {
        (char *&)obj->interface_num_in_group.data_[ i ].data_ += off;
    }
    (char *&)obj->group_interfaces_id.data_ += off;
    for( ST i = 0; i < obj->group_interfaces_id.size_; ++i ) {
        (char *&)obj->group_interfaces_id.data_[ i ].data_ += off;
    }
    (char *&)obj->side_N.data_ += off;
    for( ST i = 0; i < obj->side_N.size_; ++i ) {
        (char *&)obj->side_N.data_[ i ].data_ += off;
        for( ST j = 0; j < obj->side_N.data_[ i ].size_; ++j ) {
            (char *&)obj->side_N.data_[ i ].data_[ j ].data_ += off;
            for( ST k = 0; k < obj->side_N.data_[ i ].data_[ j ].size_; ++k ) {
                (char *&)obj->side_N.data_[ i ].data_[ j ].data_[ k ].data_ += off;
            }
        }
    }
    (char *&)obj->side_M.data_ += off;
    for( ST i = 0; i < obj->side_M.size_; ++i ) {
        (char *&)obj->side_M.data_[ i ].data_ += off;
        for( ST j = 0; j < obj->side_M.data_[ i ].size_; ++j ) {
            (char *&)obj->side_M.data_[ i ].data_[ j ].data_ += off;
        }
    }
}

void GeometryCompactClass::GroupElementsCompactClass::update_ptr_gpu_load( ST off ) {
    GeometryCompactClass__GroupElementsCompactClass__update_ptr_gpu_load<<<1,1>>>( this, off );
}

void GeometryCompactClass::GroupElementsCompactClass::update_ptr_cpu_save( ST off ) {
    for( ST i = 0; i < mesh_connectivities.size_; ++i ) {
        (char *&)mesh_connectivities.data_[ i ].data_ += off;
    }
    (char *&)mesh_connectivities.data_ += off;
    for( ST i = 0; i < connectivities.size_; ++i ) {
        (char *&)connectivities.data_[ i ].data_ += off;
    }
    (char *&)connectivities.data_ += off;
    for( ST i = 0; i < interface_group_id.size_; ++i ) {
        (char *&)interface_group_id.data_[ i ].data_ += off;
    }
    (char *&)interface_group_id.data_ += off;
    for( ST i = 0; i < interface_num_in_group.size_; ++i ) {
        (char *&)interface_num_in_group.data_[ i ].data_ += off;
    }
    (char *&)interface_num_in_group.data_ += off;
    for( ST i = 0; i < group_interfaces_id.size_; ++i ) {
        (char *&)group_interfaces_id.data_[ i ].data_ += off;
    }
    (char *&)group_interfaces_id.data_ += off;
    for( ST i = 0; i < side_N.size_; ++i ) {
        for( ST j = 0; j < side_N.data_[ i ].size_; ++j ) {
            for( ST k = 0; k < side_N.data_[ i ].data_[ j ].size_; ++k ) {
                (char *&)side_N.data_[ i ].data_[ j ].data_[ k ].data_ += off;
            }
            (char *&)side_N.data_[ i ].data_[ j ].data_ += off;
        }
        (char *&)side_N.data_[ i ].data_ += off;
    }
    (char *&)side_N.data_ += off;
    for( ST i = 0; i < side_M.size_; ++i ) {
        for( ST j = 0; j < side_M.data_[ i ].size_; ++j ) {
            (char *&)side_M.data_[ i ].data_[ j ].data_ += off;
        }
        (char *&)side_M.data_[ i ].data_ += off;
    }
    (char *&)side_M.data_ += off;
}

__global__
void GeometryCompactClass__GroupElementsCompactClass__update_ptr_gpu_save( GeometryCompactClass::GroupElementsCompactClass *obj, ST off ) {
    for( ST i = 0; i < obj->mesh_connectivities.size_; ++i ) {
        (char *&)obj->mesh_connectivities.data_[ i ].data_ += off;
    }
    (char *&)obj->mesh_connectivities.data_ += off;
    for( ST i = 0; i < obj->connectivities.size_; ++i ) {
        (char *&)obj->connectivities.data_[ i ].data_ += off;
    }
    (char *&)obj->connectivities.data_ += off;
    for( ST i = 0; i < obj->interface_group_id.size_; ++i ) {
        (char *&)obj->interface_group_id.data_[ i ].data_ += off;
    }
    (char *&)obj->interface_group_id.data_ += off;
    for( ST i = 0; i < obj->interface_num_in_group.size_; ++i ) {
        (char *&)obj->interface_num_in_group.data_[ i ].data_ += off;
    }
    (char *&)obj->interface_num_in_group.data_ += off;
    for( ST i = 0; i < obj->group_interfaces_id.size_; ++i ) {
        (char *&)obj->group_interfaces_id.data_[ i ].data_ += off;
    }
    (char *&)obj->group_interfaces_id.data_ += off;
    for( ST i = 0; i < obj->side_N.size_; ++i ) {
        for( ST j = 0; j < obj->side_N.data_[ i ].size_; ++j ) {
            for( ST k = 0; k < obj->side_N.data_[ i ].data_[ j ].size_; ++k ) {
                (char *&)obj->side_N.data_[ i ].data_[ j ].data_[ k ].data_ += off;
            }
            (char *&)obj->side_N.data_[ i ].data_[ j ].data_ += off;
        }
        (char *&)obj->side_N.data_[ i ].data_ += off;
    }
    (char *&)obj->side_N.data_ += off;
    for( ST i = 0; i < obj->side_M.size_; ++i ) {
        for( ST j = 0; j < obj->side_M.data_[ i ].size_; ++j ) {
            (char *&)obj->side_M.data_[ i ].data_[ j ].data_ += off;
        }
        (char *&)obj->side_M.data_[ i ].data_ += off;
    }
    (char *&)obj->side_M.data_ += off;
}

void GeometryCompactClass::GroupElementsCompactClass::update_ptr_gpu_save( ST off ) {
    GeometryCompactClass__GroupElementsCompactClass__update_ptr_gpu_save<<<1,1>>>( this, off );
}

void GeometryCompactClass::GroupInterfacesCompactClass::update_ptr_cpu_load( ST off ) {
    (char *&)group_elements_id.data_ += off;
    (char *&)patterns_id.data_ += off;
    (char *&)element_num_in_group.data_ += off;
    for( ST i = 0; i < element_num_in_group.size_; ++i ) {
        (char *&)element_num_in_group.data_[ i ].data_ += off;
    }
    (char *&)element_num_side.data_ += off;
    for( ST i = 0; i < element_num_side.size_; ++i ) {
        (char *&)element_num_side.data_[ i ].data_ += off;
    }
    (char *&)mesh_connectivities.data_ += off;
    for( ST i = 0; i < mesh_connectivities.size_; ++i ) {
        (char *&)mesh_connectivities.data_[ i ].data_ += off;
    }
    (char *&)connectivities.data_ += off;
    for( ST i = 0; i < connectivities.size_; ++i ) {
        (char *&)connectivities.data_[ i ].data_ += off;
    }
    (char *&)correspondance_between_nodes.data_ += off;
    for( ST i = 0; i < correspondance_between_nodes.size_; ++i ) {
        (char *&)correspondance_between_nodes.data_[ i ].data_ += off;
    }
    (char *&)Ne.data_ += off;
    for( ST i = 0; i < Ne.size_; ++i ) {
        (char *&)Ne.data_[ i ].data_ += off;
    }
    (char *&)Me.data_ += off;
}

__global__
void GeometryCompactClass__GroupInterfacesCompactClass__update_ptr_gpu_load( GeometryCompactClass::GroupInterfacesCompactClass *obj, ST off ) {
    (char *&)obj->group_elements_id.data_ += off;
    (char *&)obj->patterns_id.data_ += off;
    (char *&)obj->element_num_in_group.data_ += off;
    for( ST i = 0; i < obj->element_num_in_group.size_; ++i ) {
        (char *&)obj->element_num_in_group.data_[ i ].data_ += off;
    }
    (char *&)obj->element_num_side.data_ += off;
    for( ST i = 0; i < obj->element_num_side.size_; ++i ) {
        (char *&)obj->element_num_side.data_[ i ].data_ += off;
    }
    (char *&)obj->mesh_connectivities.data_ += off;
    for( ST i = 0; i < obj->mesh_connectivities.size_; ++i ) {
        (char *&)obj->mesh_connectivities.data_[ i ].data_ += off;
    }
    (char *&)obj->connectivities.data_ += off;
    for( ST i = 0; i < obj->connectivities.size_; ++i ) {
        (char *&)obj->connectivities.data_[ i ].data_ += off;
    }
    (char *&)obj->correspondance_between_nodes.data_ += off;
    for( ST i = 0; i < obj->correspondance_between_nodes.size_; ++i ) {
        (char *&)obj->correspondance_between_nodes.data_[ i ].data_ += off;
    }
    (char *&)obj->Ne.data_ += off;
    for( ST i = 0; i < obj->Ne.size_; ++i ) {
        (char *&)obj->Ne.data_[ i ].data_ += off;
    }
    (char *&)obj->Me.data_ += off;
}

void GeometryCompactClass::GroupInterfacesCompactClass::update_ptr_gpu_load( ST off ) {
    GeometryCompactClass__GroupInterfacesCompactClass__update_ptr_gpu_load<<<1,1>>>( this, off );
}

void GeometryCompactClass::GroupInterfacesCompactClass::update_ptr_cpu_save( ST off ) {
    (char *&)group_elements_id.data_ += off;
    (char *&)patterns_id.data_ += off;
    for( ST i = 0; i < element_num_in_group.size_; ++i ) {
        (char *&)element_num_in_group.data_[ i ].data_ += off;
    }
    (char *&)element_num_in_group.data_ += off;
    for( ST i = 0; i < element_num_side.size_; ++i ) {
        (char *&)element_num_side.data_[ i ].data_ += off;
    }
    (char *&)element_num_side.data_ += off;
    for( ST i = 0; i < mesh_connectivities.size_; ++i ) {
        (char *&)mesh_connectivities.data_[ i ].data_ += off;
    }
    (char *&)mesh_connectivities.data_ += off;
    for( ST i = 0; i < connectivities.size_; ++i ) {
        (char *&)connectivities.data_[ i ].data_ += off;
    }
    (char *&)connectivities.data_ += off;
    for( ST i = 0; i < correspondance_between_nodes.size_; ++i ) {
        (char *&)correspondance_between_nodes.data_[ i ].data_ += off;
    }
    (char *&)correspondance_between_nodes.data_ += off;
    for( ST i = 0; i < Ne.size_; ++i ) {
        (char *&)Ne.data_[ i ].data_ += off;
    }
    (char *&)Ne.data_ += off;
    (char *&)Me.data_ += off;
}

__global__
void GeometryCompactClass__GroupInterfacesCompactClass__update_ptr_gpu_save( GeometryCompactClass::GroupInterfacesCompactClass *obj, ST off ) {
    (char *&)obj->group_elements_id.data_ += off;
    (char *&)obj->patterns_id.data_ += off;
    for( ST i = 0; i < obj->element_num_in_group.size_; ++i ) {
        (char *&)obj->element_num_in_group.data_[ i ].data_ += off;
    }
    (char *&)obj->element_num_in_group.data_ += off;
    for( ST i = 0; i < obj->element_num_side.size_; ++i ) {
        (char *&)obj->element_num_side.data_[ i ].data_ += off;
    }
    (char *&)obj->element_num_side.data_ += off;
    for( ST i = 0; i < obj->mesh_connectivities.size_; ++i ) {
        (char *&)obj->mesh_connectivities.data_[ i ].data_ += off;
    }
    (char *&)obj->mesh_connectivities.data_ += off;
    for( ST i = 0; i < obj->connectivities.size_; ++i ) {
        (char *&)obj->connectivities.data_[ i ].data_ += off;
    }
    (char *&)obj->connectivities.data_ += off;
    for( ST i = 0; i < obj->correspondance_between_nodes.size_; ++i ) {
        (char *&)obj->correspondance_between_nodes.data_[ i ].data_ += off;
    }
    (char *&)obj->correspondance_between_nodes.data_ += off;
    for( ST i = 0; i < obj->Ne.size_; ++i ) {
        (char *&)obj->Ne.data_[ i ].data_ += off;
    }
    (char *&)obj->Ne.data_ += off;
    (char *&)obj->Me.data_ += off;
}

void GeometryCompactClass::GroupInterfacesCompactClass::update_ptr_gpu_save( ST off ) {
    GeometryCompactClass__GroupInterfacesCompactClass__update_ptr_gpu_save<<<1,1>>>( this, off );
}


END_METIL_NAMESPACE
