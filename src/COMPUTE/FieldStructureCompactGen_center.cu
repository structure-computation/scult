#include "hip/hip_runtime.h"
// generated file
// 
#include "FieldStructureCompactGen_center.h"

__inline__ void calculate_elements_center0(GeometryCompactClass *geometry_compact, FieldStructureCompactClass *field_structure_compact ,int i_group, int i_elem){  
        double R0 = 0.3333333333333333; double R1 = geometry_compact->nodes[0][geometry_compact->group_elements[i_group].connectivities[0][i_elem]]; double R2 = geometry_compact->nodes[0][geometry_compact->group_elements[i_group].connectivities[1][i_elem]]; double R3 = geometry_compact->nodes[0][geometry_compact->group_elements[i_group].connectivities[2][i_elem]]; double R4 = R1+R2+R3; double R5 = R0*R4; 
    field_structure_compact->group_elements[i_group].pt[0][i_elem] = R5; double R6 = geometry_compact->nodes[1][geometry_compact->group_elements[i_group].connectivities[2][i_elem]]; double R7 = geometry_compact->nodes[1][geometry_compact->group_elements[i_group].connectivities[1][i_elem]]; double R8 = geometry_compact->nodes[1][geometry_compact->group_elements[i_group].connectivities[0][i_elem]]; double R9 = R6+R7+R8; double R10 = R0*R9; 
    field_structure_compact->group_elements[i_group].pt[1][i_elem] = R10;  /* 6 instructions */
}

__global__ void global_calculate_elements_center_0(GeometryCompactClass *geometry_compact_GPU, FieldStructureCompactClass *field_structure_compact_GPU, int i_group, int i_grid, int sizeGrid, int nb_elements){
    int i_elem = i_grid * sizeGrid * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;
    if(i_elem <nb_elements){
        calculate_elements_center0(geometry_compact_GPU, field_structure_compact_GPU, i_group, i_elem);
    }
}

void global_calculate_elements_center_0_CPU(GeometryCompactClass *geometry_compact_CPU, FieldStructureCompactClass *field_structure_compact_CPU, int i_group){
    for(int i_elem=0; i_elem<geometry_compact_CPU->group_elements[i_group].nb_elements ; i_elem++)
        calculate_elements_center0(geometry_compact_CPU, field_structure_compact_CPU, i_group, i_elem);
}

__inline__ void calculate_elements_center1(GeometryCompactClass *geometry_compact, FieldStructureCompactClass *field_structure_compact ,int i_group, int i_elem){  
        double R0 = 0.4444444444444444; double R1 = geometry_compact->nodes[0][geometry_compact->group_elements[i_group].connectivities[5][i_elem]]; double R2 = geometry_compact->nodes[0][geometry_compact->group_elements[i_group].connectivities[3][i_elem]]; double R3 = geometry_compact->nodes[0][geometry_compact->group_elements[i_group].connectivities[4][i_elem]]; double R4 = R1+R2+R3; double R5 = R0*R4; 
    double R6 = -0.1111111111111111; double R7 = geometry_compact->nodes[0][geometry_compact->group_elements[i_group].connectivities[1][i_elem]]; double R8 = geometry_compact->nodes[0][geometry_compact->group_elements[i_group].connectivities[0][i_elem]]; double R9 = geometry_compact->nodes[0][geometry_compact->group_elements[i_group].connectivities[2][i_elem]]; double R10 = R7+R8+R9; double R11 = R6*R10; 
    double R12 = R5+R11; field_structure_compact->group_elements[i_group].pt[0][i_elem] = R12; double R13 = geometry_compact->nodes[1][geometry_compact->group_elements[i_group].connectivities[5][i_elem]]; double R14 = geometry_compact->nodes[1][geometry_compact->group_elements[i_group].connectivities[4][i_elem]]; double R15 = geometry_compact->nodes[1][geometry_compact->group_elements[i_group].connectivities[3][i_elem]]; double R16 = R13+R14+R15; 
    double R17 = R0*R16; double R18 = geometry_compact->nodes[1][geometry_compact->group_elements[i_group].connectivities[2][i_elem]]; double R19 = geometry_compact->nodes[1][geometry_compact->group_elements[i_group].connectivities[1][i_elem]]; double R20 = geometry_compact->nodes[1][geometry_compact->group_elements[i_group].connectivities[0][i_elem]]; double R21 = R18+R19+R20; double R22 = R6*R21; 
    double R23 = R17+R22; field_structure_compact->group_elements[i_group].pt[1][i_elem] = R23;  /* 14 instructions */
}

__global__ void global_calculate_elements_center_1(GeometryCompactClass *geometry_compact_GPU, FieldStructureCompactClass *field_structure_compact_GPU, int i_group, int i_grid, int sizeGrid, int nb_elements){
    int i_elem = i_grid * sizeGrid * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;
    if(i_elem <nb_elements){
        calculate_elements_center1(geometry_compact_GPU, field_structure_compact_GPU, i_group, i_elem);
    }
}

void global_calculate_elements_center_1_CPU(GeometryCompactClass *geometry_compact_CPU, FieldStructureCompactClass *field_structure_compact_CPU, int i_group){
    for(int i_elem=0; i_elem<geometry_compact_CPU->group_elements[i_group].nb_elements ; i_elem++)
        calculate_elements_center1(geometry_compact_CPU, field_structure_compact_CPU, i_group, i_elem);
}

__inline__ void calculate_elements_center2(GeometryCompactClass *geometry_compact, FieldStructureCompactClass *field_structure_compact ,int i_group, int i_elem){  
        double R0 = 0.25; double R1 = geometry_compact->nodes[0][geometry_compact->group_elements[i_group].connectivities[0][i_elem]]; double R2 = geometry_compact->nodes[0][geometry_compact->group_elements[i_group].connectivities[2][i_elem]]; double R3 = geometry_compact->nodes[0][geometry_compact->group_elements[i_group].connectivities[1][i_elem]]; double R4 = geometry_compact->nodes[0][geometry_compact->group_elements[i_group].connectivities[3][i_elem]]; double R5 = R1+R2+R3+R4; 
    double R6 = R0*R5; field_structure_compact->group_elements[i_group].pt[0][i_elem] = R6; double R7 = geometry_compact->nodes[1][geometry_compact->group_elements[i_group].connectivities[3][i_elem]]; double R8 = geometry_compact->nodes[1][geometry_compact->group_elements[i_group].connectivities[0][i_elem]]; double R9 = geometry_compact->nodes[1][geometry_compact->group_elements[i_group].connectivities[1][i_elem]]; double R10 = geometry_compact->nodes[1][geometry_compact->group_elements[i_group].connectivities[2][i_elem]]; 
    double R11 = R7+R8+R9+R10; double R12 = R0*R11; field_structure_compact->group_elements[i_group].pt[1][i_elem] = R12; double R13 = geometry_compact->nodes[2][geometry_compact->group_elements[i_group].connectivities[3][i_elem]]; double R14 = geometry_compact->nodes[2][geometry_compact->group_elements[i_group].connectivities[0][i_elem]]; double R15 = geometry_compact->nodes[2][geometry_compact->group_elements[i_group].connectivities[1][i_elem]]; 
    double R16 = geometry_compact->nodes[2][geometry_compact->group_elements[i_group].connectivities[2][i_elem]]; double R17 = R13+R14+R15+R16; double R18 = R0*R17; field_structure_compact->group_elements[i_group].pt[2][i_elem] = R18;  /* 12 instructions */
}

__global__ void global_calculate_elements_center_2(GeometryCompactClass *geometry_compact_GPU, FieldStructureCompactClass *field_structure_compact_GPU, int i_group, int i_grid, int sizeGrid, int nb_elements){
    int i_elem = i_grid * sizeGrid * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;
    if(i_elem <nb_elements){
        calculate_elements_center2(geometry_compact_GPU, field_structure_compact_GPU, i_group, i_elem);
    }
}

void global_calculate_elements_center_2_CPU(GeometryCompactClass *geometry_compact_CPU, FieldStructureCompactClass *field_structure_compact_CPU, int i_group){
    for(int i_elem=0; i_elem<geometry_compact_CPU->group_elements[i_group].nb_elements ; i_elem++)
        calculate_elements_center2(geometry_compact_CPU, field_structure_compact_CPU, i_group, i_elem);
}

__inline__ void calculate_elements_center3(GeometryCompactClass *geometry_compact, FieldStructureCompactClass *field_structure_compact ,int i_group, int i_elem){  
        double R0 = -0.125; double R1 = geometry_compact->nodes[0][geometry_compact->group_elements[i_group].connectivities[3][i_elem]]; double R2 = geometry_compact->nodes[0][geometry_compact->group_elements[i_group].connectivities[0][i_elem]]; double R3 = geometry_compact->nodes[0][geometry_compact->group_elements[i_group].connectivities[1][i_elem]]; double R4 = geometry_compact->nodes[0][geometry_compact->group_elements[i_group].connectivities[2][i_elem]]; double R5 = R1+R2+R3+R4; 
    double R6 = R0*R5; double R7 = 0.25; double R8 = geometry_compact->nodes[0][geometry_compact->group_elements[i_group].connectivities[4][i_elem]]; double R9 = geometry_compact->nodes[0][geometry_compact->group_elements[i_group].connectivities[9][i_elem]]; double R10 = geometry_compact->nodes[0][geometry_compact->group_elements[i_group].connectivities[8][i_elem]]; double R11 = geometry_compact->nodes[0][geometry_compact->group_elements[i_group].connectivities[7][i_elem]]; 
    double R12 = geometry_compact->nodes[0][geometry_compact->group_elements[i_group].connectivities[5][i_elem]]; double R13 = geometry_compact->nodes[0][geometry_compact->group_elements[i_group].connectivities[6][i_elem]]; double R14 = R8+R9+R10+R11+R12+R13; double R15 = R7*R14; double R16 = R6+R15; field_structure_compact->group_elements[i_group].pt[0][i_elem] = R16; 
    double R17 = geometry_compact->nodes[1][geometry_compact->group_elements[i_group].connectivities[3][i_elem]]; double R18 = geometry_compact->nodes[1][geometry_compact->group_elements[i_group].connectivities[0][i_elem]]; double R19 = geometry_compact->nodes[1][geometry_compact->group_elements[i_group].connectivities[2][i_elem]]; double R20 = geometry_compact->nodes[1][geometry_compact->group_elements[i_group].connectivities[1][i_elem]]; double R21 = R17+R18+R19+R20; double R22 = R0*R21; 
    double R23 = geometry_compact->nodes[1][geometry_compact->group_elements[i_group].connectivities[9][i_elem]]; double R24 = geometry_compact->nodes[1][geometry_compact->group_elements[i_group].connectivities[8][i_elem]]; double R25 = geometry_compact->nodes[1][geometry_compact->group_elements[i_group].connectivities[7][i_elem]]; double R26 = geometry_compact->nodes[1][geometry_compact->group_elements[i_group].connectivities[6][i_elem]]; double R27 = geometry_compact->nodes[1][geometry_compact->group_elements[i_group].connectivities[4][i_elem]]; double R28 = geometry_compact->nodes[1][geometry_compact->group_elements[i_group].connectivities[5][i_elem]]; 
    double R29 = R23+R24+R25+R26+R27+R28; double R30 = R7*R29; double R31 = R22+R30; field_structure_compact->group_elements[i_group].pt[1][i_elem] = R31; double R32 = geometry_compact->nodes[2][geometry_compact->group_elements[i_group].connectivities[3][i_elem]]; double R33 = geometry_compact->nodes[2][geometry_compact->group_elements[i_group].connectivities[2][i_elem]]; 
    double R34 = geometry_compact->nodes[2][geometry_compact->group_elements[i_group].connectivities[1][i_elem]]; double R35 = geometry_compact->nodes[2][geometry_compact->group_elements[i_group].connectivities[0][i_elem]]; double R36 = R32+R33+R34+R35; double R37 = R0*R36; double R38 = geometry_compact->nodes[2][geometry_compact->group_elements[i_group].connectivities[9][i_elem]]; double R39 = geometry_compact->nodes[2][geometry_compact->group_elements[i_group].connectivities[8][i_elem]]; 
    double R40 = geometry_compact->nodes[2][geometry_compact->group_elements[i_group].connectivities[7][i_elem]]; double R41 = geometry_compact->nodes[2][geometry_compact->group_elements[i_group].connectivities[6][i_elem]]; double R42 = geometry_compact->nodes[2][geometry_compact->group_elements[i_group].connectivities[4][i_elem]]; double R43 = geometry_compact->nodes[2][geometry_compact->group_elements[i_group].connectivities[5][i_elem]]; double R44 = R38+R39+R40+R41+R42+R43; double R45 = R7*R44; 
    double R46 = R37+R45; field_structure_compact->group_elements[i_group].pt[2][i_elem] = R46;  /* 33 instructions */
}

__global__ void global_calculate_elements_center_3(GeometryCompactClass *geometry_compact_GPU, FieldStructureCompactClass *field_structure_compact_GPU, int i_group, int i_grid, int sizeGrid, int nb_elements){
    int i_elem = i_grid * sizeGrid * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;
    if(i_elem <nb_elements){
        calculate_elements_center3(geometry_compact_GPU, field_structure_compact_GPU, i_group, i_elem);
    }
}

void global_calculate_elements_center_3_CPU(GeometryCompactClass *geometry_compact_CPU, FieldStructureCompactClass *field_structure_compact_CPU, int i_group){
    for(int i_elem=0; i_elem<geometry_compact_CPU->group_elements[i_group].nb_elements ; i_elem++)
        calculate_elements_center3(geometry_compact_CPU, field_structure_compact_CPU, i_group, i_elem);
}

