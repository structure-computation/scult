#include "hip/hip_runtime.h"
#include "FieldStructureCompactClass.h"

BEG_METIL_NAMESPACE

void FieldStructureCompactClass::update_ptr_cpu_load( ST off ) {
    (char *&)patterns.types.data_ += off;
    for( ST i = 0; i < patterns.types.size_; ++i ) {
        (char *&)patterns.types.data_[ i ].permutation.data_ += off;
        (char *&)patterns.types.data_[ i ].sides.data_ += off;
        for( ST j = 0; j < patterns.types.data_[ i ].sides.size_; ++j ) {
            (char *&)patterns.types.data_[ i ].sides.data_[ j ].data_ += off;
        }
    }
    (char *&)group_elements.data_ += off;
    for( ST i = 0; i < group_elements.size_; ++i ) {
        (char *&)group_elements.data_[ i ].pt.data_ += off;
        for( ST j = 0; j < group_elements.data_[ i ].pt.size_; ++j ) {
            (char *&)group_elements.data_[ i ].pt.data_[ j ].data_ += off;
        }
        (char *&)group_elements.data_[ i ].mat_prop.data_ += off;
        for( ST j = 0; j < group_elements.data_[ i ].mat_prop.size_; ++j ) {
            (char *&)group_elements.data_[ i ].mat_prop.data_[ j ].data_ += off;
        }
        (char *&)group_elements.data_[ i ].volumic_force.data_ += off;
        for( ST j = 0; j < group_elements.data_[ i ].volumic_force.size_; ++j ) {
            (char *&)group_elements.data_[ i ].volumic_force.data_[ j ].data_ += off;
        }
        (char *&)group_elements.data_[ i ].mat_elem.data_ += off;
        for( ST j = 0; j < group_elements.data_[ i ].mat_elem.size_; ++j ) {
            (char *&)group_elements.data_[ i ].mat_elem.data_[ j ].data_ += off;
        }
        (char *&)group_elements.data_[ i ].size.data_ += off;
    }
    (char *&)group_interfaces.data_ += off;
    for( ST i = 0; i < group_interfaces.size_; ++i ) {
        (char *&)group_interfaces.data_[ i ].pt.data_ += off;
        for( ST j = 0; j < group_interfaces.data_[ i ].pt.size_; ++j ) {
            (char *&)group_interfaces.data_[ i ].pt.data_[ j ].data_ += off;
        }
        (char *&)group_interfaces.data_[ i ].link_prop.data_ += off;
        for( ST j = 0; j < group_interfaces.data_[ i ].link_prop.size_; ++j ) {
            (char *&)group_interfaces.data_[ i ].link_prop.data_[ j ].data_ += off;
        }
        (char *&)group_interfaces.data_[ i ].BC_step_prop.data_ += off;
        for( ST j = 0; j < group_interfaces.data_[ i ].BC_step_prop.size_; ++j ) {
            (char *&)group_interfaces.data_[ i ].BC_step_prop.data_[ j ].data_ += off;
        }
        (char *&)group_interfaces.data_[ i ].bc.data_ += off;
        for( ST j = 0; j < group_interfaces.data_[ i ].bc.size_; ++j ) {
            (char *&)group_interfaces.data_[ i ].bc.data_[ j ].data_ += off;
        }
    }
}

__global__
void FieldStructureCompactClass__update_ptr_gpu_load( FieldStructureCompactClass *obj, ST off ) {
    (char *&)obj->patterns.types.data_ += off;
    for( ST i = 0; i < obj->patterns.types.size_; ++i ) {
        (char *&)obj->patterns.types.data_[ i ].permutation.data_ += off;
        (char *&)obj->patterns.types.data_[ i ].sides.data_ += off;
        for( ST j = 0; j < obj->patterns.types.data_[ i ].sides.size_; ++j ) {
            (char *&)obj->patterns.types.data_[ i ].sides.data_[ j ].data_ += off;
        }
    }
    (char *&)obj->group_elements.data_ += off;
    for( ST i = 0; i < obj->group_elements.size_; ++i ) {
        (char *&)obj->group_elements.data_[ i ].pt.data_ += off;
        for( ST j = 0; j < obj->group_elements.data_[ i ].pt.size_; ++j ) {
            (char *&)obj->group_elements.data_[ i ].pt.data_[ j ].data_ += off;
        }
        (char *&)obj->group_elements.data_[ i ].mat_prop.data_ += off;
        for( ST j = 0; j < obj->group_elements.data_[ i ].mat_prop.size_; ++j ) {
            (char *&)obj->group_elements.data_[ i ].mat_prop.data_[ j ].data_ += off;
        }
        (char *&)obj->group_elements.data_[ i ].volumic_force.data_ += off;
        for( ST j = 0; j < obj->group_elements.data_[ i ].volumic_force.size_; ++j ) {
            (char *&)obj->group_elements.data_[ i ].volumic_force.data_[ j ].data_ += off;
        }
        (char *&)obj->group_elements.data_[ i ].mat_elem.data_ += off;
        for( ST j = 0; j < obj->group_elements.data_[ i ].mat_elem.size_; ++j ) {
            (char *&)obj->group_elements.data_[ i ].mat_elem.data_[ j ].data_ += off;
        }
        (char *&)obj->group_elements.data_[ i ].size.data_ += off;
    }
    (char *&)obj->group_interfaces.data_ += off;
    for( ST i = 0; i < obj->group_interfaces.size_; ++i ) {
        (char *&)obj->group_interfaces.data_[ i ].pt.data_ += off;
        for( ST j = 0; j < obj->group_interfaces.data_[ i ].pt.size_; ++j ) {
            (char *&)obj->group_interfaces.data_[ i ].pt.data_[ j ].data_ += off;
        }
        (char *&)obj->group_interfaces.data_[ i ].link_prop.data_ += off;
        for( ST j = 0; j < obj->group_interfaces.data_[ i ].link_prop.size_; ++j ) {
            (char *&)obj->group_interfaces.data_[ i ].link_prop.data_[ j ].data_ += off;
        }
        (char *&)obj->group_interfaces.data_[ i ].BC_step_prop.data_ += off;
        for( ST j = 0; j < obj->group_interfaces.data_[ i ].BC_step_prop.size_; ++j ) {
            (char *&)obj->group_interfaces.data_[ i ].BC_step_prop.data_[ j ].data_ += off;
        }
        (char *&)obj->group_interfaces.data_[ i ].bc.data_ += off;
        for( ST j = 0; j < obj->group_interfaces.data_[ i ].bc.size_; ++j ) {
            (char *&)obj->group_interfaces.data_[ i ].bc.data_[ j ].data_ += off;
        }
    }
}

void FieldStructureCompactClass::update_ptr_gpu_load( ST off ) {
    FieldStructureCompactClass__update_ptr_gpu_load<<<1,1>>>( this, off );
}

void FieldStructureCompactClass::update_ptr_cpu_save( ST off ) {
    for( ST i = 0; i < patterns.types.size_; ++i ) {
        (char *&)patterns.types.data_[ i ].permutation.data_ += off;
        for( ST j = 0; j < patterns.types.data_[ i ].sides.size_; ++j ) {
            (char *&)patterns.types.data_[ i ].sides.data_[ j ].data_ += off;
        }
        (char *&)patterns.types.data_[ i ].sides.data_ += off;
    }
    (char *&)patterns.types.data_ += off;
    for( ST i = 0; i < group_elements.size_; ++i ) {
        for( ST j = 0; j < group_elements.data_[ i ].pt.size_; ++j ) {
            (char *&)group_elements.data_[ i ].pt.data_[ j ].data_ += off;
        }
        (char *&)group_elements.data_[ i ].pt.data_ += off;
        for( ST j = 0; j < group_elements.data_[ i ].mat_prop.size_; ++j ) {
            (char *&)group_elements.data_[ i ].mat_prop.data_[ j ].data_ += off;
        }
        (char *&)group_elements.data_[ i ].mat_prop.data_ += off;
        for( ST j = 0; j < group_elements.data_[ i ].volumic_force.size_; ++j ) {
            (char *&)group_elements.data_[ i ].volumic_force.data_[ j ].data_ += off;
        }
        (char *&)group_elements.data_[ i ].volumic_force.data_ += off;
        for( ST j = 0; j < group_elements.data_[ i ].mat_elem.size_; ++j ) {
            (char *&)group_elements.data_[ i ].mat_elem.data_[ j ].data_ += off;
        }
        (char *&)group_elements.data_[ i ].mat_elem.data_ += off;
        (char *&)group_elements.data_[ i ].size.data_ += off;
    }
    (char *&)group_elements.data_ += off;
    for( ST i = 0; i < group_interfaces.size_; ++i ) {
        for( ST j = 0; j < group_interfaces.data_[ i ].pt.size_; ++j ) {
            (char *&)group_interfaces.data_[ i ].pt.data_[ j ].data_ += off;
        }
        (char *&)group_interfaces.data_[ i ].pt.data_ += off;
        for( ST j = 0; j < group_interfaces.data_[ i ].link_prop.size_; ++j ) {
            (char *&)group_interfaces.data_[ i ].link_prop.data_[ j ].data_ += off;
        }
        (char *&)group_interfaces.data_[ i ].link_prop.data_ += off;
        for( ST j = 0; j < group_interfaces.data_[ i ].BC_step_prop.size_; ++j ) {
            (char *&)group_interfaces.data_[ i ].BC_step_prop.data_[ j ].data_ += off;
        }
        (char *&)group_interfaces.data_[ i ].BC_step_prop.data_ += off;
        for( ST j = 0; j < group_interfaces.data_[ i ].bc.size_; ++j ) {
            (char *&)group_interfaces.data_[ i ].bc.data_[ j ].data_ += off;
        }
        (char *&)group_interfaces.data_[ i ].bc.data_ += off;
    }
    (char *&)group_interfaces.data_ += off;
}

__global__
void FieldStructureCompactClass__update_ptr_gpu_save( FieldStructureCompactClass *obj, ST off ) {
    for( ST i = 0; i < obj->patterns.types.size_; ++i ) {
        (char *&)obj->patterns.types.data_[ i ].permutation.data_ += off;
        for( ST j = 0; j < obj->patterns.types.data_[ i ].sides.size_; ++j ) {
            (char *&)obj->patterns.types.data_[ i ].sides.data_[ j ].data_ += off;
        }
        (char *&)obj->patterns.types.data_[ i ].sides.data_ += off;
    }
    (char *&)obj->patterns.types.data_ += off;
    for( ST i = 0; i < obj->group_elements.size_; ++i ) {
        for( ST j = 0; j < obj->group_elements.data_[ i ].pt.size_; ++j ) {
            (char *&)obj->group_elements.data_[ i ].pt.data_[ j ].data_ += off;
        }
        (char *&)obj->group_elements.data_[ i ].pt.data_ += off;
        for( ST j = 0; j < obj->group_elements.data_[ i ].mat_prop.size_; ++j ) {
            (char *&)obj->group_elements.data_[ i ].mat_prop.data_[ j ].data_ += off;
        }
        (char *&)obj->group_elements.data_[ i ].mat_prop.data_ += off;
        for( ST j = 0; j < obj->group_elements.data_[ i ].volumic_force.size_; ++j ) {
            (char *&)obj->group_elements.data_[ i ].volumic_force.data_[ j ].data_ += off;
        }
        (char *&)obj->group_elements.data_[ i ].volumic_force.data_ += off;
        for( ST j = 0; j < obj->group_elements.data_[ i ].mat_elem.size_; ++j ) {
            (char *&)obj->group_elements.data_[ i ].mat_elem.data_[ j ].data_ += off;
        }
        (char *&)obj->group_elements.data_[ i ].mat_elem.data_ += off;
        (char *&)obj->group_elements.data_[ i ].size.data_ += off;
    }
    (char *&)obj->group_elements.data_ += off;
    for( ST i = 0; i < obj->group_interfaces.size_; ++i ) {
        for( ST j = 0; j < obj->group_interfaces.data_[ i ].pt.size_; ++j ) {
            (char *&)obj->group_interfaces.data_[ i ].pt.data_[ j ].data_ += off;
        }
        (char *&)obj->group_interfaces.data_[ i ].pt.data_ += off;
        for( ST j = 0; j < obj->group_interfaces.data_[ i ].link_prop.size_; ++j ) {
            (char *&)obj->group_interfaces.data_[ i ].link_prop.data_[ j ].data_ += off;
        }
        (char *&)obj->group_interfaces.data_[ i ].link_prop.data_ += off;
        for( ST j = 0; j < obj->group_interfaces.data_[ i ].BC_step_prop.size_; ++j ) {
            (char *&)obj->group_interfaces.data_[ i ].BC_step_prop.data_[ j ].data_ += off;
        }
        (char *&)obj->group_interfaces.data_[ i ].BC_step_prop.data_ += off;
        for( ST j = 0; j < obj->group_interfaces.data_[ i ].bc.size_; ++j ) {
            (char *&)obj->group_interfaces.data_[ i ].bc.data_[ j ].data_ += off;
        }
        (char *&)obj->group_interfaces.data_[ i ].bc.data_ += off;
    }
    (char *&)obj->group_interfaces.data_ += off;
}

void FieldStructureCompactClass::update_ptr_gpu_save( ST off ) {
    FieldStructureCompactClass__update_ptr_gpu_save<<<1,1>>>( this, off );
}

void FieldStructureCompactClass::Patterns::update_ptr_cpu_load( ST off ) {
    (char *&)types.data_ += off;
    for( ST i = 0; i < types.size_; ++i ) {
        (char *&)types.data_[ i ].permutation.data_ += off;
        (char *&)types.data_[ i ].sides.data_ += off;
        for( ST j = 0; j < types.data_[ i ].sides.size_; ++j ) {
            (char *&)types.data_[ i ].sides.data_[ j ].data_ += off;
        }
    }
}

__global__
void FieldStructureCompactClass__Patterns__update_ptr_gpu_load( FieldStructureCompactClass::Patterns *obj, ST off ) {
    (char *&)obj->types.data_ += off;
    for( ST i = 0; i < obj->types.size_; ++i ) {
        (char *&)obj->types.data_[ i ].permutation.data_ += off;
        (char *&)obj->types.data_[ i ].sides.data_ += off;
        for( ST j = 0; j < obj->types.data_[ i ].sides.size_; ++j ) {
            (char *&)obj->types.data_[ i ].sides.data_[ j ].data_ += off;
        }
    }
}

void FieldStructureCompactClass::Patterns::update_ptr_gpu_load( ST off ) {
    FieldStructureCompactClass__Patterns__update_ptr_gpu_load<<<1,1>>>( this, off );
}

void FieldStructureCompactClass::Patterns::update_ptr_cpu_save( ST off ) {
    for( ST i = 0; i < types.size_; ++i ) {
        (char *&)types.data_[ i ].permutation.data_ += off;
        for( ST j = 0; j < types.data_[ i ].sides.size_; ++j ) {
            (char *&)types.data_[ i ].sides.data_[ j ].data_ += off;
        }
        (char *&)types.data_[ i ].sides.data_ += off;
    }
    (char *&)types.data_ += off;
}

__global__
void FieldStructureCompactClass__Patterns__update_ptr_gpu_save( FieldStructureCompactClass::Patterns *obj, ST off ) {
    for( ST i = 0; i < obj->types.size_; ++i ) {
        (char *&)obj->types.data_[ i ].permutation.data_ += off;
        for( ST j = 0; j < obj->types.data_[ i ].sides.size_; ++j ) {
            (char *&)obj->types.data_[ i ].sides.data_[ j ].data_ += off;
        }
        (char *&)obj->types.data_[ i ].sides.data_ += off;
    }
    (char *&)obj->types.data_ += off;
}

void FieldStructureCompactClass::Patterns::update_ptr_gpu_save( ST off ) {
    FieldStructureCompactClass__Patterns__update_ptr_gpu_save<<<1,1>>>( this, off );
}

void FieldStructureCompactClass::Patterns::Types::update_ptr_cpu_load( ST off ) {
    (char *&)permutation.data_ += off;
    (char *&)sides.data_ += off;
    for( ST i = 0; i < sides.size_; ++i ) {
        (char *&)sides.data_[ i ].data_ += off;
    }
}

__global__
void FieldStructureCompactClass__Patterns__Types__update_ptr_gpu_load( FieldStructureCompactClass::Patterns::Types *obj, ST off ) {
    (char *&)obj->permutation.data_ += off;
    (char *&)obj->sides.data_ += off;
    for( ST i = 0; i < obj->sides.size_; ++i ) {
        (char *&)obj->sides.data_[ i ].data_ += off;
    }
}

void FieldStructureCompactClass::Patterns::Types::update_ptr_gpu_load( ST off ) {
    FieldStructureCompactClass__Patterns__Types__update_ptr_gpu_load<<<1,1>>>( this, off );
}

void FieldStructureCompactClass::Patterns::Types::update_ptr_cpu_save( ST off ) {
    (char *&)permutation.data_ += off;
    for( ST i = 0; i < sides.size_; ++i ) {
        (char *&)sides.data_[ i ].data_ += off;
    }
    (char *&)sides.data_ += off;
}

__global__
void FieldStructureCompactClass__Patterns__Types__update_ptr_gpu_save( FieldStructureCompactClass::Patterns::Types *obj, ST off ) {
    (char *&)obj->permutation.data_ += off;
    for( ST i = 0; i < obj->sides.size_; ++i ) {
        (char *&)obj->sides.data_[ i ].data_ += off;
    }
    (char *&)obj->sides.data_ += off;
}

void FieldStructureCompactClass::Patterns::Types::update_ptr_gpu_save( ST off ) {
    FieldStructureCompactClass__Patterns__Types__update_ptr_gpu_save<<<1,1>>>( this, off );
}

void FieldStructureCompactClass::GroupFieldStructureElements::update_ptr_cpu_load( ST off ) {
    (char *&)pt.data_ += off;
    for( ST i = 0; i < pt.size_; ++i ) {
        (char *&)pt.data_[ i ].data_ += off;
    }
    (char *&)mat_prop.data_ += off;
    for( ST i = 0; i < mat_prop.size_; ++i ) {
        (char *&)mat_prop.data_[ i ].data_ += off;
    }
    (char *&)volumic_force.data_ += off;
    for( ST i = 0; i < volumic_force.size_; ++i ) {
        (char *&)volumic_force.data_[ i ].data_ += off;
    }
    (char *&)mat_elem.data_ += off;
    for( ST i = 0; i < mat_elem.size_; ++i ) {
        (char *&)mat_elem.data_[ i ].data_ += off;
    }
    (char *&)size.data_ += off;
}

__global__
void FieldStructureCompactClass__GroupFieldStructureElements__update_ptr_gpu_load( FieldStructureCompactClass::GroupFieldStructureElements *obj, ST off ) {
    (char *&)obj->pt.data_ += off;
    for( ST i = 0; i < obj->pt.size_; ++i ) {
        (char *&)obj->pt.data_[ i ].data_ += off;
    }
    (char *&)obj->mat_prop.data_ += off;
    for( ST i = 0; i < obj->mat_prop.size_; ++i ) {
        (char *&)obj->mat_prop.data_[ i ].data_ += off;
    }
    (char *&)obj->volumic_force.data_ += off;
    for( ST i = 0; i < obj->volumic_force.size_; ++i ) {
        (char *&)obj->volumic_force.data_[ i ].data_ += off;
    }
    (char *&)obj->mat_elem.data_ += off;
    for( ST i = 0; i < obj->mat_elem.size_; ++i ) {
        (char *&)obj->mat_elem.data_[ i ].data_ += off;
    }
    (char *&)obj->size.data_ += off;
}

void FieldStructureCompactClass::GroupFieldStructureElements::update_ptr_gpu_load( ST off ) {
    FieldStructureCompactClass__GroupFieldStructureElements__update_ptr_gpu_load<<<1,1>>>( this, off );
}

void FieldStructureCompactClass::GroupFieldStructureElements::update_ptr_cpu_save( ST off ) {
    for( ST i = 0; i < pt.size_; ++i ) {
        (char *&)pt.data_[ i ].data_ += off;
    }
    (char *&)pt.data_ += off;
    for( ST i = 0; i < mat_prop.size_; ++i ) {
        (char *&)mat_prop.data_[ i ].data_ += off;
    }
    (char *&)mat_prop.data_ += off;
    for( ST i = 0; i < volumic_force.size_; ++i ) {
        (char *&)volumic_force.data_[ i ].data_ += off;
    }
    (char *&)volumic_force.data_ += off;
    for( ST i = 0; i < mat_elem.size_; ++i ) {
        (char *&)mat_elem.data_[ i ].data_ += off;
    }
    (char *&)mat_elem.data_ += off;
    (char *&)size.data_ += off;
}

__global__
void FieldStructureCompactClass__GroupFieldStructureElements__update_ptr_gpu_save( FieldStructureCompactClass::GroupFieldStructureElements *obj, ST off ) {
    for( ST i = 0; i < obj->pt.size_; ++i ) {
        (char *&)obj->pt.data_[ i ].data_ += off;
    }
    (char *&)obj->pt.data_ += off;
    for( ST i = 0; i < obj->mat_prop.size_; ++i ) {
        (char *&)obj->mat_prop.data_[ i ].data_ += off;
    }
    (char *&)obj->mat_prop.data_ += off;
    for( ST i = 0; i < obj->volumic_force.size_; ++i ) {
        (char *&)obj->volumic_force.data_[ i ].data_ += off;
    }
    (char *&)obj->volumic_force.data_ += off;
    for( ST i = 0; i < obj->mat_elem.size_; ++i ) {
        (char *&)obj->mat_elem.data_[ i ].data_ += off;
    }
    (char *&)obj->mat_elem.data_ += off;
    (char *&)obj->size.data_ += off;
}

void FieldStructureCompactClass::GroupFieldStructureElements::update_ptr_gpu_save( ST off ) {
    FieldStructureCompactClass__GroupFieldStructureElements__update_ptr_gpu_save<<<1,1>>>( this, off );
}

void FieldStructureCompactClass::GroupFieldStructureInterfaces::update_ptr_cpu_load( ST off ) {
    (char *&)pt.data_ += off;
    for( ST i = 0; i < pt.size_; ++i ) {
        (char *&)pt.data_[ i ].data_ += off;
    }
    (char *&)link_prop.data_ += off;
    for( ST i = 0; i < link_prop.size_; ++i ) {
        (char *&)link_prop.data_[ i ].data_ += off;
    }
    (char *&)BC_step_prop.data_ += off;
    for( ST i = 0; i < BC_step_prop.size_; ++i ) {
        (char *&)BC_step_prop.data_[ i ].data_ += off;
    }
    (char *&)bc.data_ += off;
    for( ST i = 0; i < bc.size_; ++i ) {
        (char *&)bc.data_[ i ].data_ += off;
    }
}

__global__
void FieldStructureCompactClass__GroupFieldStructureInterfaces__update_ptr_gpu_load( FieldStructureCompactClass::GroupFieldStructureInterfaces *obj, ST off ) {
    (char *&)obj->pt.data_ += off;
    for( ST i = 0; i < obj->pt.size_; ++i ) {
        (char *&)obj->pt.data_[ i ].data_ += off;
    }
    (char *&)obj->link_prop.data_ += off;
    for( ST i = 0; i < obj->link_prop.size_; ++i ) {
        (char *&)obj->link_prop.data_[ i ].data_ += off;
    }
    (char *&)obj->BC_step_prop.data_ += off;
    for( ST i = 0; i < obj->BC_step_prop.size_; ++i ) {
        (char *&)obj->BC_step_prop.data_[ i ].data_ += off;
    }
    (char *&)obj->bc.data_ += off;
    for( ST i = 0; i < obj->bc.size_; ++i ) {
        (char *&)obj->bc.data_[ i ].data_ += off;
    }
}

void FieldStructureCompactClass::GroupFieldStructureInterfaces::update_ptr_gpu_load( ST off ) {
    FieldStructureCompactClass__GroupFieldStructureInterfaces__update_ptr_gpu_load<<<1,1>>>( this, off );
}

void FieldStructureCompactClass::GroupFieldStructureInterfaces::update_ptr_cpu_save( ST off ) {
    for( ST i = 0; i < pt.size_; ++i ) {
        (char *&)pt.data_[ i ].data_ += off;
    }
    (char *&)pt.data_ += off;
    for( ST i = 0; i < link_prop.size_; ++i ) {
        (char *&)link_prop.data_[ i ].data_ += off;
    }
    (char *&)link_prop.data_ += off;
    for( ST i = 0; i < BC_step_prop.size_; ++i ) {
        (char *&)BC_step_prop.data_[ i ].data_ += off;
    }
    (char *&)BC_step_prop.data_ += off;
    for( ST i = 0; i < bc.size_; ++i ) {
        (char *&)bc.data_[ i ].data_ += off;
    }
    (char *&)bc.data_ += off;
}

__global__
void FieldStructureCompactClass__GroupFieldStructureInterfaces__update_ptr_gpu_save( FieldStructureCompactClass::GroupFieldStructureInterfaces *obj, ST off ) {
    for( ST i = 0; i < obj->pt.size_; ++i ) {
        (char *&)obj->pt.data_[ i ].data_ += off;
    }
    (char *&)obj->pt.data_ += off;
    for( ST i = 0; i < obj->link_prop.size_; ++i ) {
        (char *&)obj->link_prop.data_[ i ].data_ += off;
    }
    (char *&)obj->link_prop.data_ += off;
    for( ST i = 0; i < obj->BC_step_prop.size_; ++i ) {
        (char *&)obj->BC_step_prop.data_[ i ].data_ += off;
    }
    (char *&)obj->BC_step_prop.data_ += off;
    for( ST i = 0; i < obj->bc.size_; ++i ) {
        (char *&)obj->bc.data_[ i ].data_ += off;
    }
    (char *&)obj->bc.data_ += off;
}

void FieldStructureCompactClass::GroupFieldStructureInterfaces::update_ptr_gpu_save( ST off ) {
    FieldStructureCompactClass__GroupFieldStructureInterfaces__update_ptr_gpu_save<<<1,1>>>( this, off );
}


END_METIL_NAMESPACE
