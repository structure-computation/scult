#include "hip/hip_runtime.h"
// generated file
// 
#include "FieldStructureCompactGen.h"

void launch_calculate_elements_center_GPU(Ps<GeometryCompactClass> geometry_compact_CPU, Ps<GeometryCompactClass> geometry_compact_GPU, Ps<FieldStructureCompactClass> field_structure_compact_GPU){
    for(int i_group=0; i_group < geometry_compact_CPU->group_elements.size(); i_group++){
        int threadsPerBlock = THREADSPERBLOCK;
        int sizeGrid = SIZEGRID;
        int nGrids = std::floor( geometry_compact_CPU->group_elements[i_group].connectivities[0].size()  / (sizeGrid*threadsPerBlock) ) + 1 ;  // obtenir l'entier superieur
        int lastGridSize = (geometry_compact_CPU->group_elements[i_group].connectivities[0].size() - (sizeGrid*threadsPerBlock)*(nGrids-1))/threadsPerBlock;
        switch (geometry_compact_CPU->group_elements[i_group].pattern_id){
            //for pattern PATTERN_Triangle
            case 0 :{
                for(int i_grid=0; i_grid < nGrids; i_grid++){
                    int blocksPerGrid;
                    if(i_grid == nGrids-1){
                        blocksPerGrid = lastGridSize;
                    }else{
                        blocksPerGrid = sizeGrid;
                    }
                    global_calculate_elements_center_0<<<blocksPerGrid,threadsPerBlock>>>(geometry_compact_GPU.ptr(), field_structure_compact_GPU.ptr(), i_group, i_grid, sizeGrid, geometry_compact_CPU->group_elements[i_group].nb_elements);
                }
                break;
            }
            //for pattern PATTERN_Triangle_6
            case 1 :{
                for(int i_grid=0; i_grid < nGrids; i_grid++){
                    int blocksPerGrid;
                    if(i_grid == nGrids-1){
                        blocksPerGrid = lastGridSize;
                    }else{
                        blocksPerGrid = sizeGrid;
                    }
                    global_calculate_elements_center_1<<<blocksPerGrid,threadsPerBlock>>>(geometry_compact_GPU.ptr(), field_structure_compact_GPU.ptr(), i_group, i_grid, sizeGrid, geometry_compact_CPU->group_elements[i_group].nb_elements);
                }
                break;
            }
            //for pattern PATTERN_Tetra
            case 2 :{
                for(int i_grid=0; i_grid < nGrids; i_grid++){
                    int blocksPerGrid;
                    if(i_grid == nGrids-1){
                        blocksPerGrid = lastGridSize;
                    }else{
                        blocksPerGrid = sizeGrid;
                    }
                    global_calculate_elements_center_2<<<blocksPerGrid,threadsPerBlock>>>(geometry_compact_GPU.ptr(), field_structure_compact_GPU.ptr(), i_group, i_grid, sizeGrid, geometry_compact_CPU->group_elements[i_group].nb_elements);
                }
                break;
            }
            //for pattern PATTERN_Tetra_10
            case 3 :{
                for(int i_grid=0; i_grid < nGrids; i_grid++){
                    int blocksPerGrid;
                    if(i_grid == nGrids-1){
                        blocksPerGrid = lastGridSize;
                    }else{
                        blocksPerGrid = sizeGrid;
                    }
                    global_calculate_elements_center_3<<<blocksPerGrid,threadsPerBlock>>>(geometry_compact_GPU.ptr(), field_structure_compact_GPU.ptr(), i_group, i_grid, sizeGrid, geometry_compact_CPU->group_elements[i_group].nb_elements);
                }
                break;
            }
            default :{
                std::cerr << "type de pattern non implemente" << std::endl; assert(0);                    
            }
        }
    }
}

void launch_calculate_elements_center_CPU(Ps<GeometryCompactClass> geometry_compact_CPU, Ps<FieldStructureCompactClass> field_structure_compact_CPU){
    for(int i_group=0; i_group < geometry_compact_CPU->group_elements.size(); i_group++){
        switch (geometry_compact_CPU->group_elements[i_group].pattern_id){
            //for pattern PATTERN_Triangle
            case 0 :{
                global_calculate_elements_center_0_CPU(geometry_compact_CPU.ptr(), field_structure_compact_CPU.ptr(), i_group);
                break;
            }
            //for pattern PATTERN_Triangle_6
            case 1 :{
                global_calculate_elements_center_1_CPU(geometry_compact_CPU.ptr(), field_structure_compact_CPU.ptr(), i_group);
                break;
            }
            //for pattern PATTERN_Tetra
            case 2 :{
                global_calculate_elements_center_2_CPU(geometry_compact_CPU.ptr(), field_structure_compact_CPU.ptr(), i_group);
                break;
            }
            //for pattern PATTERN_Tetra_10
            case 3 :{
                global_calculate_elements_center_3_CPU(geometry_compact_CPU.ptr(), field_structure_compact_CPU.ptr(), i_group);
                break;
            }
            default :{
                std::cerr << "type de pattern non implemente" << std::endl; assert(0);
            }
            
        }
    }
}

// void launch_calculate_elements_mat_elem_GPU(Ps<GeometryCompactClass> geometry_compact_CPU, Ps<GeometryCompactClass> geometry_compact_GPU, Ps<FieldStructureCompactClass> field_structure_compact_GPU){
//     for(int i_group=0; i_group < geometry_compact_CPU->group_elements.size(); i_group++){
//         int threadsPerBlock = THREADSPERBLOCK;
//         int sizeGrid = SIZEGRID;
//         int nGrids = std::floor( geometry_compact_CPU->group_elements[i_group].connectivities[0].size()  / (sizeGrid*threadsPerBlock) ) + 1 ;  // obtenir l'entier superieur
//         int lastGridSize = (geometry_compact_CPU->group_elements[i_group].connectivities[0].size() - (sizeGrid*threadsPerBlock)*(nGrids-1))/threadsPerBlock;
//         switch (geometry_compact_CPU->group_elements[i_group].pattern_id){
//             //for pattern PATTERN_Triangle
//             case 0 :{
//                 for(int i_grid=0; i_grid < nGrids; i_grid++){
//                     int blocksPerGrid;
//                     if(i_grid == nGrids-1){
//                         blocksPerGrid = lastGridSize;
//                     }else{
//                         blocksPerGrid = sizeGrid;
//                     }
//                     global_calculate_elements_mat_elem_0<<<blocksPerGrid,threadsPerBlock>>>(geometry_compact_GPU.ptr(), field_structure_compact_GPU.ptr(), i_group, i_grid, sizeGrid, geometry_compact_CPU->group_elements[i_group].nb_elements);
//                 }
//                 break;
//             }
//             //for pattern PATTERN_Triangle_6
//             case 1 :{
//                 for(int i_grid=0; i_grid < nGrids; i_grid++){
//                     int blocksPerGrid;
//                     if(i_grid == nGrids-1){
//                         blocksPerGrid = lastGridSize;
//                     }else{
//                         blocksPerGrid = sizeGrid;
//                     }
//                     global_calculate_elements_mat_elem_1<<<blocksPerGrid,threadsPerBlock>>>(geometry_compact_GPU.ptr(), field_structure_compact_GPU.ptr(), i_group, i_grid, sizeGrid, geometry_compact_CPU->group_elements[i_group].nb_elements);
//                 }
//                 break;
//             }
//             //for pattern PATTERN_Tetra
//             case 2 :{
//                 for(int i_grid=0; i_grid < nGrids; i_grid++){
//                     int blocksPerGrid;
//                     if(i_grid == nGrids-1){
//                         blocksPerGrid = lastGridSize;
//                     }else{
//                         blocksPerGrid = sizeGrid;
//                     }
//                     global_calculate_elements_mat_elem_2<<<blocksPerGrid,threadsPerBlock>>>(geometry_compact_GPU.ptr(), field_structure_compact_GPU.ptr(), i_group, i_grid, sizeGrid, geometry_compact_CPU->group_elements[i_group].nb_elements);
//                 }
//                 break;
//             }
//             //for pattern PATTERN_Tetra_10
//             case 3 :{
//                 for(int i_grid=0; i_grid < nGrids; i_grid++){
//                     int blocksPerGrid;
//                     if(i_grid == nGrids-1){
//                         blocksPerGrid = lastGridSize;
//                     }else{
//                         blocksPerGrid = sizeGrid;
//                     }
//                     global_calculate_elements_mat_elem_3<<<blocksPerGrid,threadsPerBlock>>>(geometry_compact_GPU.ptr(), field_structure_compact_GPU.ptr(), i_group, i_grid, sizeGrid, geometry_compact_CPU->group_elements[i_group].nb_elements);
//                 }
//                 break;
//             }
//             default :{
//                 std::cerr << "type de pattern non implemente" << std::endl; assert(0);                    
//             }
//         }
//     }
// }
// 
// void launch_calculate_elements_mat_elem_CPU(Ps<GeometryCompactClass> geometry_compact_CPU, Ps<FieldStructureCompactClass> field_structure_compact_CPU){
//     for(int i_group=0; i_group < geometry_compact_CPU->group_elements.size(); i_group++){
//         switch (geometry_compact_CPU->group_elements[i_group].pattern_id){
//             //for pattern PATTERN_Triangle
//             case 0 :{
//                 global_calculate_elements_mat_elem_0_CPU(geometry_compact_CPU.ptr(), field_structure_compact_CPU.ptr(), i_group);
//                 break;
//             }
//             //for pattern PATTERN_Triangle_6
//             case 1 :{
//                 global_calculate_elements_mat_elem_1_CPU(geometry_compact_CPU.ptr(), field_structure_compact_CPU.ptr(), i_group);
//                 break;
//             }
//             //for pattern PATTERN_Tetra
//             case 2 :{
//                 global_calculate_elements_mat_elem_2_CPU(geometry_compact_CPU.ptr(), field_structure_compact_CPU.ptr(), i_group);
//                 break;
//             }
//             //for pattern PATTERN_Tetra_10
//             case 3 :{
//                 global_calculate_elements_mat_elem_3_CPU(geometry_compact_CPU.ptr(), field_structure_compact_CPU.ptr(), i_group);
//                 break;
//             }
//             default :{
//                 std::cerr << "type de pattern non implemente" << std::endl; assert(0);
//             }
//             
//         }
//     }
// }
// 
